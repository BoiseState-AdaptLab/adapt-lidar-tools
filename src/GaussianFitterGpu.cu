#include "hip/hip_runtime.h"
#include "GaussianFitterGpu.cuh"

__global__
void func_df_kernel(size_t npeaks, size_t vec_stride, double* vec_data,
                    size_t data_size, double* data_t, 
                    size_t mat_cols, double* matrix) {
    //indexes i (index for data->t[i]) and j (index for gsl_vector)
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    //verify both i and j are within bounds to calculate sums of derivatives
    if (i < data_size && j < npeaks) {
        //get values from vector since gsl is not ported to gpu
        //equivalent to gsl_vector_get(v, 3*j+0)
        double a = vec_data[(3*j + 0) * vec_stride];
        double b = vec_data[(3*j + 1) * vec_stride];
        double c = vec_data[(3*j + 2) * vec_stride];

        //get data->t[i]
        double ti = data_t[i];
        double zi = (ti - b) / c;
        double ei = exp(-0.5 * zi * zi);
        
        //calculate sums of derivatives
        double a_sum = (-1)*ei;
        double b_sum = (-1)*a*(ti-b)*ei*(1/(c*c));
        double c_sum = (-1)*a*(ti-b)*(ti-b) * ei * (1/(c*c*c)); 

        //set values in matrix since gsl set matrix is not ported to gpu
        //gsl_matrix_set(J, i,3*j+ 0, a_sum);
        matrix[i*mat_cols + (3*j+0)] = a_sum;

        //gsl_matrix_set(J, i,3*j+ 1, b_sum);
        matrix[i*mat_cols + (3*j+1)] = b_sum;

        //gsl_matrix_set(J, i,3*j+ 1, c_sum);
        matrix[i*mat_cols + (3*j+2)] = c_sum;
    } 
}

double* array_double_alloc(double* host_arr, size_t host_size) {
    double* device_arr;

    // allocate memory for struct on device (gpu)
    hipMalloc(&device_arr, sizeof(double)*host_size);

    hipMemcpy(device_arr, host_arr, sizeof(double)*host_size, hipMemcpyHostToDevice);

    return device_arr;
}

void func_df_gpu(const gsl_vector* x, double* data_t, size_t data_t_size, gsl_matrix* J) {
    double* device_vec;
    double* device_data_t;
    double* device_matrix;
    size_t npeaks = x->size/3;

    device_vec = array_double_alloc(x->data, x->size);
    device_data_t = array_double_alloc(data_t, data_t_size);

    hipMalloc(&device_matrix, J->tda*J->size1*sizeof(double));

    //call kernel
    dim3 dimBlock(16,16);
    dim3 dimGrid((data_t_size + dimBlock.x - 1)/dimBlock.x, 
                   (npeaks + dimBlock.y - 1)/dimBlock.y);

    func_df_kernel<<<dimGrid, dimBlock>>>(npeaks, x->stride, device_vec, 
                                          data_t_size, device_data_t, 
                                          J->tda, device_matrix);

    //sync: wait until gpu threads finish
    hipDeviceSynchronize();

    //copy output matrix back to host
    hipMemcpy(J->data, device_matrix, J->size1*J->tda*sizeof(double), hipMemcpyDeviceToHost);

    //clean up gpu memory
    hipFree(device_vec);
    hipFree(device_data_t);
    hipFree(device_matrix);
}

